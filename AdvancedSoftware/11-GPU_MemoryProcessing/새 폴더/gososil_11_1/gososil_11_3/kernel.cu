#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include <time.h>
#include <assert.h>
#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#define N 20
#define BLOCK_SIZE 4

float* A, * B, * C;
float * X0, *X1, *FX0, *FX1;

__global__ void roots_kernel(float *a, float* b, float* c,
	float* phi1, float* phi2, float *xx, float * xx2)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	float a2, b2, c2, d2, x02, x12, tmp2;

	a2 = a[i]; b2 = b[i]; c2 = c[i];
	d2 = sqrtf(b2 * b2 - 4.0f * a2 * c2);
	tmp2 = 1.0f / (2.0f * a2);
	phi1[i] = x02 = (-b2 - d2) * tmp2;
	phi2[i] = x12 = (-b2 + d2) * tmp2;
	xx[i] = (a2 * x02 + b2) * x02 + c2;
	xx2[i] = (a2 * x12 + b2) * x12 + c2;
}

void find_roots_CPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n) {
	int i;
	float a, b, c, d, x0, x1, tmp;
	for (i = 0; i < n; i++) {
		a = A[i]; b = B[i]; c = C[i];
		d = sqrtf(b * b - 4.0f * a * c);
		tmp = 1.0f / (2.0f * a);
		X0[i] = x0 = (-b - d) * tmp;
		X1[i] = x1 = (-b + d) * tmp;
		FX0[i] = (a * x0 + b) * x0 + c;
		FX1[i] = (a * x1 + b) * x1 + c;
	}
}


void find_roots_GPU(float* A, float* B, float* C,
	float* X0, float* X1, float* FX0, float* FX1, int n)
{
	//�Ʒ� �Լ����� ����Ͽ� ��� �ϸ� ������ ��Ȯ�� �ð��� ������ �� ������ �����غ� ��.
	//CHECK_TIME_INIT_GPU()
	//	CHECK_TIME_START_GPU()
	//	CHECK_TIME_END_GPU(device_time)
	//	CHECK_TIME_DEST_GPU()

		hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}/////////////  if(cu.....  ==CUDA_CALL

	size_t size;
	float* a2, * b2, * c2, *x0, *x1, *fx0, *fx1;

	size = n* sizeof(float);
	CUDA_CALL(hipMalloc(&a2, size))
		CUDA_CALL(hipMemcpy(a2, A, size, hipMemcpyHostToDevice))
		CUDA_CALL(hipMalloc(&b2, size))
		CUDA_CALL(hipMemcpy(b2, B, size, hipMemcpyHostToDevice))
		CUDA_CALL(hipMalloc(&c2, size))
		CUDA_CALL(hipMemcpy(c2, C, size, hipMemcpyHostToDevice))

	CUDA_CALL(hipMalloc(&x0, size))
		CUDA_CALL(hipMalloc(&x1, size))
		CUDA_CALL(hipMalloc(&fx0, size))
		CUDA_CALL(hipMalloc(&fx1, size))

		//    // Assume that width and height are multiples of BLOCK SIZE.
			//������ ���� �ϴ� ��
			//�� �ȿ� �����尡 ��������(=32)��ŭ ���� (�����尡 ������ŭ ������)
		dim3 dimBlock(BLOCK_SIZE); // dimblock(block_size, 1, 1) ��� �ȿ� 32���� �����尡 ����
	//���� �� ����. ��� �ϳ��� �����尡 blocksize��ŭ ����.
	//������Ʈ ����� �� ������ ����, �����.x == ����� ����
	dim3 dimGrid(N / dimBlock.x); //��, ����� �Ѱ����� ������Ʈ������/�����(=32)
	roots_kernel << < dimGrid, dimBlock >> > (a2, b2, c2, x0, x1, fx0, fx1);

	CUDA_CALL(hipGetLastError())

		//    // hipDeviceSynchronize waits for the kernel to finish, and returns
		//    // any errors encountered during the launch.
		CUDA_CALL(hipDeviceSynchronize())
		///*CUDA_CALL(hipMemcpy(p_VecY_G, p_VecY_G2, size, hipMemcpyDeviceToHost))
		//CUDA_CALL(hipMemcpy(p_VecY_G, p_VecY_G2, size, hipMemcpyDeviceToHost))*/

		Error:
	//hipFree(p_VecY_G2);
	//hipFree(p_VecX2);
	//hipFree(p_MatA2);

}

int main()
{

	A = new float[20]; B = new float[20]; C = new float[20];
	X0 = new float[20]; X1 = new float[20]; FX0 = new float[20]; FX1 = new float[20];
	float a, b, c;
	srand((unsigned)time(NULL));
	for (int i = 0; i < 20; i++) {
		a =  ((float)rand() / RAND_MAX) * 10.0f;
		b =  ((float)rand() / RAND_MAX) * 10.0f;
		c =  ((float)rand() / RAND_MAX) * 10.0f;

		if (b * b - 4 * a * c >= 0) {
			A[i] = a; B[i] = b; C[i] = c;
		}
		else
			i--;
	}
	find_roots_CPU(A, B, C, X0, X1, FX0, FX1, N);

	for(int i=0; i<20; i++)
		printf("%f %f %f %f %f %f %f\n", A[i], B[i], C[i], X0[i], X1[i], FX0[i], FX1[i]);

	find_roots_GPU(A, B, C, X0, X1, FX0, FX1, N);

	printf("\n");
	for (int i = 0; i < 20; i++)
		printf("%f %f %f %f %f %f %f\n", A[i], B[i], C[i], X0[i], X1[i], FX0[i], FX1[i]);

	return 0;
}