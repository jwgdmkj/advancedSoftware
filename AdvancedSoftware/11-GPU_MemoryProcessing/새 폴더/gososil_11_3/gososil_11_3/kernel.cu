#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#define	IN
#define OUT
#define INOUT

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define N_SIZE (1 << 3)													// ��ü ������ ������
#define NF_SIZE (1 << 2)													// Nf ũ��

#define BLOCK_SIZE (1 << 3)													// CUDA Ŀ�� thread block ������

#define BLOCK_WIDTH (1 << 3)
#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)

#define N_ITERATION (1 << 0)												// ���� �ݺ� Ƚ��

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int Nf;

int *h_ArrayElements;
int *h_SumOfArrayElements_CPU;
int *h_SumOfArrayElements_GPU;

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, 
    OUT int *p_SumOfElements_GPU, int Nf);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� Ŀ�� �ڵ�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Sum_n_elements_Kernel(IN int *d_ArrayElements,
    OUT int *d_SumOfArrayElements, int N, int Nf) {
    /*Todo*/
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int sum = d_ArrayElements[Nf-1];

    for (int x = 1; x <= idx; x++) {
        if ((Nf-1 - x) < 0 &&
            (Nf-1 + x) > N) {
            continue;
        }

        else if ((Nf-1 - x) < 0) {
            sum += d_ArrayElements[Nf-1 + x];
        }
        else if ((Nf-1 + x) > N) {
            sum += d_ArrayElements[Nf-1 - x];
        }

        else {
            sum += d_ArrayElements[Nf-1 - x] + d_ArrayElements[Nf-1 + x];
        }
    }

    d_SumOfArrayElements[idx] = sum;
    printf("%d ", sum);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� C �ڵ�
//	GPU kernel�� ����� �񱳸� ���� ���� ����� �Ͽ����� �Ǵ��ϴ� �����ͷ� Ȱ��
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Sum_n_elements_CPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_CPU, int Nf) {
    /*Todo*/

    int sum;
    printf("%d\n", N);
    for (int i = 0; i < N; i++) {
        sum = p_ArrayElements[Nf-1];

        for (int x = 1; x <= i; x++) {
            if ((Nf-1 - x) < 0 &&
                (Nf-1 + x) > _msize(p_ArrayElements) / sizeof(int)) {
                continue;
            }

            else if ((Nf-1 - x) < 0) {
                sum += p_ArrayElements[Nf-1 + x];
            }
            else if ((Nf-1 + x) > _msize(p_ArrayElements) / sizeof(int)) {
                sum += p_ArrayElements[Nf-1 - x];
            }

            else {
                sum += p_ArrayElements[Nf-1 - x] + p_ArrayElements[Nf-1 + x];
            }
        }

        p_SumOfElements_CPU[i] = sum;
        printf("%d ", sum);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�־��� bin ������ �д� �ڵ�
//	ù 4����Ʈ�� ��ü �������� ����, ���� 4����Ʈ�� Nf�� ũ��, �� ���� N���� int�� �����Ͱ� ����
//	�����ʹ� -100 ~ 100 ������ ���� ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void read_bin_file() {
    printf("***Binary File Read Start!!\n");
    FILE *fp = fopen("gen.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(&Nf, sizeof(int), 1, fp);

    h_ArrayElements = (int *)malloc(N * sizeof(int));
    h_SumOfArrayElements_CPU = (int *)malloc(N * sizeof(int));
    h_SumOfArrayElements_GPU = (int *)malloc(N * sizeof(int));

    fread(h_ArrayElements, sizeof(int), N, fp);

    fclose(fp);
    printf("***Binary File Read End!!\n\n");
}

void init_bin_file(IN int n, IN int nf) {
    printf("***Binary File Create Start!!\n");
    srand((unsigned)time(NULL));
    FILE *fp = fopen("gen.bin", "wb");
    fwrite(&n, sizeof(int), 1, fp);
    fwrite(&nf, sizeof(int), 1, fp);

    int i, input;

    for (i = 0; i < n; i++) {
        input = (int)((float)rand() / RAND_MAX * 200 - 100);
        fwrite(&input, sizeof(int), 1, fp);
    }

    fclose(fp);
    printf("***Binary File Create End!!\n\n");
}

int main()
{
    int i;
    init_bin_file(N_SIZE, NF_SIZE);
    read_bin_file();

    TIMER_T CPU_time = 0.0f, GPU_time_NO_SHARED = 0.0f;
    printf("%d -> %d\n", Nf, h_ArrayElements[Nf-1]);
    printf("%d & %d\n", h_ArrayElements[Nf-2], h_ArrayElements[Nf]);
    printf("%d & %d\n", h_ArrayElements[Nf -3], h_ArrayElements[Nf+1]);
    for (i = 0; i < N_ITERATION; i++) {
        CHECK_TIME_START;
        Sum_n_elements_CPU(h_ArrayElements, h_SumOfArrayElements_CPU, Nf);
        CHECK_TIME_END(compute_time);
        CPU_time += compute_time;
        printf("\n");
        Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU, Nf);
        GPU_time_NO_SHARED += device_time;
    }

    for (i = 0; i < N; i++) {
        if (h_SumOfArrayElements_CPU[i] != h_SumOfArrayElements_GPU[i]) {
            printf("%d : CPU : %d,\tGPU : %d\n", i, h_SumOfArrayElements_CPU[i], h_SumOfArrayElements_GPU[i]);
            break;
        }
    }
    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    printf("***CPU compute time : %.3f ms\n", CPU_time / N_ITERATION);
    printf("***GPU NO SHARED compute time : %.3f ms\n", GPU_time_NO_SHARED / N_ITERATION);

    free(h_ArrayElements);
    free(h_SumOfArrayElements_CPU);
    free(h_SumOfArrayElements_GPU);

    return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, 
    int Nf) {
    hipError_t cudaStatus;
    /*Todo*/
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
                goto Error;
    }

    int* p_ArrayElements2, * p_SumOfElements_GPU2;
    size_t size;
    size = N * sizeof(int);
    CUDA_CALL(hipMalloc(&p_ArrayElements2, size))
        CUDA_CALL(hipMemcpy(p_ArrayElements2, p_ArrayElements, size, hipMemcpyHostToDevice))

        CUDA_CALL(hipMalloc(&p_SumOfElements_GPU2, size))

        //    // Assume that width and height are multiples of BLOCK SIZE.
            //������ ���� �ϴ� ��
            //�� �ȿ� �����尡 ��������(=32)��ŭ ���� (�����尡 ������ŭ ������)
        dim3 dimBlock(BLOCK_SIZE); // dimblock(block_size, 1, 1) ��� �ȿ� 32���� �����尡 ����
    //���� �� ����. ��� �ϳ��� �����尡 blocksize��ŭ ����.
    //������Ʈ ����� �� ������ ����, �����.x == ����� ����
    dim3 dimGrid(N_SIZE / dimBlock.x); //��, ����� �Ѱ����� ������Ʈ������/�����(=32)
    Sum_n_elements_Kernel << < dimGrid, dimBlock >> > (p_ArrayElements2,
        p_SumOfElements_GPU2,N, Nf);

    CUDA_CALL(hipGetLastError())

        //    // hipDeviceSynchronize waits for the kernel to finish, and returns
        //    // any errors encountered during the launch.
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(p_SumOfElements_GPU, p_SumOfElements_GPU2, size, hipMemcpyDeviceToHost))

        CHECK_TIME_DEST_GPU();

Error:
    hipFree(p_ArrayElements2);
    hipFree(p_SumOfElements_GPU2);


    return cudaStatus;
}
