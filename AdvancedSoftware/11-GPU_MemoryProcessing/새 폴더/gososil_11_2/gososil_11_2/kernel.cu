#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 32
#define DATA_SIZE (1 << 26)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int* h_Fibonacci_number;
int* h_fibonacci_value;
int* h_fibonacci_value_GPU;

hipError_t Fibonacci_GPU(int* h_Fibo_n, int* h_Fibo_v, int* h_Fibo_v_GPU);

__global__ void Fibonacci_Kernel(int* x, int* y)
{
    /*Thread Idx*/
    int row;
    int col;
    int tid;

    /*Todo*/
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float sqrt_5, x_0, x_1, tmp_0, tmp_1;

    sqrt_5 = sqrtf(5.0f);
    x_0 = (1.0f + sqrt_5) / 2.0f;
    x_1 = (1.0f - sqrt_5) / 2.0f;

    tmp_0 = tmp_1 = 1.0f;

    for (int j = 0; j < x[i]; j++) {
        tmp_0 *= x_0;
        tmp_1 *= x_1;
    }
    y[i] = (int)((tmp_0 - tmp_1) / sqrt_5 + 0.5f);
}


void Fibonacci_CPU(int* h_Fibo_n, int* h_Fibo_v) {
    int i, j;

    float sqrt_5, x_0, x_1, tmp_0, tmp_1;

    sqrt_5 = sqrtf(5.0f);
    x_0 = (1.0f + sqrt_5) / 2.0f;
    x_1 = (1.0f - sqrt_5) / 2.0f;

    for (i = 0; i < N; i++) {
        tmp_0 = tmp_1 = 1.0f;
        for (j = 0; j < h_Fibo_n[i]; j++) {
            tmp_0 *= x_0;
            tmp_1 *= x_1;
        }
        h_Fibo_v[i] = (int)((tmp_0 - tmp_1) / sqrt_5 + 0.5f);
    }

}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
    FILE* fp = fopen("x.bin", "wb");

    fwrite(&size, sizeof(int), 1, fp);

    int random_num;
    for (int i = 0; i < size; i++) {
        random_num = (int)(((double)rand() / RAND_MAX) * 40.0f);
        fwrite(&random_num, sizeof(int), 1, fp);
    }

    fclose(fp);
    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE* fp = fopen("x.bin", "rb");

    /*Todo*/
    int random_num;
    fread(&N, sizeof(int), 1, fp);

    h_Fibonacci_number = new int[N];
    h_fibonacci_value = new int[N];
    h_fibonacci_value_GPU = new int[N];

    for (int i = 0; i < N; i++) {
        fread(&random_num, sizeof(int), 1, fp);
        h_Fibonacci_number[i] = random_num;
    }

    fclose(fp);
    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file(DATA_SIZE);
    read_bin_file();

    printf("The problem size is %d.\n", N);
    int i;
    FILE* fp;

    //CPU 
    printf("***Fibonacci_CPU Start!!\n");
    CHECK_TIME_START;
    Fibonacci_CPU(h_Fibonacci_number, h_fibonacci_value);
    CHECK_TIME_END(compute_time);
    printf("***Fibonacci_CPU End!!\n\n");

    printf("CPU time = %.6f\n", compute_time);
    //GPU
    printf("%d\n", N);
    printf("***Fibonacci_GPU Start!!\n");
    Fibonacci_GPU(h_Fibonacci_number, h_fibonacci_value, h_fibonacci_value_GPU);
    printf("***Fibonacci_GPU End!!\n\n");

    printf("GPU time = %.6f\n", device_time);

    for (i = 0; i < N; i++)
    {
        if (h_fibonacci_value[i] != h_fibonacci_value_GPU[i])
        {
            printf("%u != %u\n", h_fibonacci_value[i], h_fibonacci_value_GPU[i]);
            break;
        }
    }

    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    // Write the output array into the output file.
    fp = fopen("y.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N, sizeof(int), 1, fp);

    for (i = 0; i < N; i++)
        fwrite(&h_fibonacci_value_GPU[i], sizeof(int), 1, fp);

    fclose(fp);
    printf("end!!\n\n");

    return 0;
}


hipError_t Fibonacci_GPU(int* h_Fibo_n, int* h_Fibo_v, int* h_Fibo_v_GPU)
{
    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU()
        CHECK_TIME_END_GPU(device_time)
        CHECK_TIME_DEST_GPU()

        hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        //        goto Error;
    }
    /*Todo*/
    int* x, * y;
    size_t size;
    size = N * sizeof(int);
    CUDA_CALL(hipMalloc(&x, size))
        CUDA_CALL(hipMemcpy(x, h_Fibo_n, size, hipMemcpyHostToDevice))

        CUDA_CALL(hipMalloc(&y, size))

        //    // Assume that width and height are multiples of BLOCK SIZE.
            //������ ���� �ϴ� ��
            //�� �ȿ� �����尡 ��������(=32)��ŭ ���� (�����尡 ������ŭ ������)
        dim3 dimBlock(BLOCK_SIZE); // dimblock(block_size, 1, 1) ��� �ȿ� 32���� �����尡 ����
    //���� �� ����. ��� �ϳ��� �����尡 blocksize��ŭ ����.
    //������Ʈ ����� �� ������ ����, �����.x == ����� ����
    dim3 dimGrid(DATA_SIZE / dimBlock.x); //��, ����� �Ѱ����� ������Ʈ������/�����(=32)
    Fibonacci_Kernel << < dimGrid, dimBlock >> > (x, y);

    CUDA_CALL(hipGetLastError())

        //    // hipDeviceSynchronize waits for the kernel to finish, and returns
        //    // any errors encountered during the launch.
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(h_Fibo_v_GPU, y, size, hipMemcpyDeviceToHost))

        CHECK_TIME_DEST_GPU();
    return cudaStatus;
}
