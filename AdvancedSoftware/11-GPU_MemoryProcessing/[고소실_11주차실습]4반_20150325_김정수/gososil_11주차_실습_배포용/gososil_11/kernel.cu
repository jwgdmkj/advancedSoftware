#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

//1 prac 2
//2 prac 3
#define prac 2


#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1
#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#if prac==1



typedef struct {
    int width;
    int height;
    float *elements;
} Array;


#define MAX_N_ELEMENTS	(1 << 20)

void generate_random_float_array(float *array, int n) {

    int i;

    for (i = 0; i < n; i++) {
        array[i] = 3.1415926f*((float)rand() / RAND_MAX);
    }
}
void combine_two_arrays(float *x, float *y, float *z, int n) {
    /*int i;*/

    for (i = 0; i < n; i++) {
        z[i] = 1.0f / (sin(x[i])*cos(y[i]) + cos(x[i])*sin(y[i]));
    }
}

__global__ void CombineTwoArrraysKernel(Array A, Array B, Array C) {
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    int id = gridDim.x*blockDim.x*row + col;
    C.elements[id] = 1.0f / (sin(A.elements[id])*cos(B.elements[id]) +
        cos(A.elements[id])*sin(B.elements[id]));
}

hipError_t combine_two_arrays_GPU(const Array A, const Array B, Array C);

int BLOCK_SIZE = 16;

int main()
{
    int n_elements;

    srand((unsigned int)time(NULL));
    n_elements = MAX_N_ELEMENTS;
    Array A, B, C, G;
    A.width = B.width = C.width = G.width = 1024;
    A.height = B.height = C.height = G.height = MAX_N_ELEMENTS / 1024;

    A.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
    B.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
    C.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
    G.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
    generate_random_float_array(A.elements, MAX_N_ELEMENTS);
    generate_random_float_array(B.elements, MAX_N_ELEMENTS);

    CHECK_TIME_START;
    combine_two_arrays(A.elements, B.elements, C.elements, n_elements);
    CHECK_TIME_END(compute_time);

    printf("***GPU C[10] = %f/ Time taken = %.6fms\n", C.elements[10], compute_time);


    hipError_t cudaStatus = combine_two_arrays_GPU(A, B, G);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "combine_two_arrays_GPU failed!");
        return 1;
    }
    printf("***GPU G[10] = %f/ Time taken = %.6fms\n", G.elements[10], device_time);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
hipError_t combine_two_arrays_GPU(const Array A, const Array B, Array C) {

    //�Ʒ� �Լ����� ����Ͽ� ��� �ϸ� ������ ��Ȯ�� �ð��� ������ �� ������ �����غ� ��.
    CHECK_TIME_INIT_GPU()
        CHECK_TIME_START_GPU()
        CHECK_TIME_END_GPU(device_time)
        CHECK_TIME_DEST_GPU()

        hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }/////////////  if(cu.....  ==CUDA_CALL

    Array d_A, d_B, d_C;
    size_t size;

    d_A.width = A.width; d_A.height = A.height;
    size = A.width * A.height * sizeof(float);
    CUDA_CALL(hipMalloc(&d_A.elements, size))

        CUDA_CALL(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice))

        d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    CUDA_CALL(hipMalloc(&d_B.elements, size))
        CUDA_CALL(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice))

        d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    CUDA_CALL(hipMalloc(&d_C.elements, size))

        // Assume that width and height are multiples of BLOCK SIZE.
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(A.width / dimBlock.x, A.height / dimBlock.y);
    CombineTwoArrraysKernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C);

    CUDA_CALL(hipGetLastError())

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost))


        Error:
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
    return cudaStatus;
}
#endif

#if prac==2

int n;
#define BLOCK_SIZE 32
#define ELEMENT_SIZE (1<<10)
const int ELEM_PER_VECTOR = 32;
float(*pVecX), (*pVecY), (*pVecY_G);
float(*pMatA);
void init_MatVec(void)
{
    srand((unsigned)time(NULL));
    FILE* fp = fopen("gen.bin", "rb");
    fread(&n, sizeof(float), 1, fp);

    pVecX = new float[n * ELEM_PER_VECTOR];
    pVecY = new float[n * ELEM_PER_VECTOR];
    pVecY_G = new float[n * ELEM_PER_VECTOR];
    pMatA = new float[ELEM_PER_VECTOR * ELEM_PER_VECTOR];

    fread(pVecX, sizeof(float), n * ELEM_PER_VECTOR, fp);
    fread(pMatA, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);
    fclose(fp);
}
void Mat_Vec_Multiply()
{
    int vec_idx, i, j;

    for (vec_idx = 0; vec_idx < ELEMENT_SIZE; vec_idx++) {
        for (i = 0; i < ELEM_PER_VECTOR; i++) {
            float sum = 0;
            for (j = 0; j < ELEM_PER_VECTOR; j++) {
                sum += pMatA[i * ELEM_PER_VECTOR + j] * pVecX[vec_idx * ELEM_PER_VECTOR + j];
            }
            pVecY[vec_idx * ELEM_PER_VECTOR + i] = sum;
        }
    }
}

__global__ void Mat_Vec_Multiply_Kernel(float *d_VecY, float *d_VecX,
    float *d_MatA, int Vec_Size)
{
    /*int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    int id = gridDim.x*blockDim.x*row + col;
    d_MatA.elements[id] = 1.0f / (sin(A.elements[id])*cos(B.elements[id]) +
        cos(A.elements[id])*sin(B.elements[id]));*/

    int i, j;
    int thr_idx = blockDim.x * blockIdx.x + threadIdx.x;

    for (i = 0; i < ELEM_PER_VECTOR; i++) {
        float sum = 0;
        for (j = 0; j < ELEM_PER_VECTOR; j++) {
            sum += d_MatA[i * ELEM_PER_VECTOR + j] * d_VecX[thr_idx * ELEM_PER_VECTOR + j];
        }
        d_VecY[thr_idx * ELEM_PER_VECTOR + i] = sum;
    }
}

void Mat_Vec_Multiply_GPU(float *p_VecX, float *p_MatA, float *p_VecY_G)
{
    //�Ʒ� �Լ����� ����Ͽ� ��� �ϸ� ������ ��Ȯ�� �ð��� ������ �� ������ �����غ� ��.
    CHECK_TIME_INIT_GPU()
        CHECK_TIME_START_GPU()
        CHECK_TIME_END_GPU(device_time)
        CHECK_TIME_DEST_GPU()

        hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }/////////////  if(cu.....  ==CUDA_CALL

  //  Array d_A, d_B, d_C;
    size_t size;
    float * p_VecX2, * p_MatA2, * p_VecY_G2;
 //   d_A = new float[size * (_msize(p_VecX) / sizeof(float))];
 //   d_B = new float[_msize(p_VecX) / sizeof(float)];
  //  d_C = new float[ELEM_PER_VECTOR * ELEM_PER_VECTOR];

 //   d_A.width = ELEM_PER_VECTOR; d_A.height = ELEM_PR_VECTOR;
    size = ELEM_PER_VECTOR * ELEM_PER_VECTOR * sizeof(float);
    CUDA_CALL(hipMalloc(&p_MatA2, size))
    CUDA_CALL(hipMemcpy(p_MatA2, pMatA, size, hipMemcpyHostToDevice))

//        d_B.width = ELEM_PER_VECTOR; d_B.height = ELEM_PER_VECTOR;
    size = n * ELEM_PER_VECTOR * sizeof(float);
    CUDA_CALL(hipMalloc(&p_VecX2, size))
        CUDA_CALL(hipMemcpy(p_VecX2, p_VecX, size, hipMemcpyHostToDevice))

  //      d_C.width = ELEM_PER_VECTOR; d_C.height = ELEM_PER_VECTOR;
    size =n * ELEM_PER_VECTOR * sizeof(float);
    CUDA_CALL(hipMalloc(&p_VecY_G2, size))

    //    // Assume that width and height are multiples of BLOCK SIZE.
        //������ ���� �ϴ� ��
        //�� �ȿ� �����尡 ��������(=32)��ŭ ���� (�����尡 ������ŭ ������)
        dim3 dimBlock(BLOCK_SIZE); // dimblock(block_size, 1, 1) ��� �ȿ� 32���� �����尡 ����
    //���� �� ����. ��� �ϳ��� �����尡 blocksize��ŭ ����.
    //������Ʈ ����� �� ������ ����, �����.x == ����� ����
    dim3 dimGrid(ELEMENT_SIZE/dimBlock.x); //��, ����� �Ѱ����� ������Ʈ������/�����(=32)
    Mat_Vec_Multiply_Kernel << < dimGrid, dimBlock >> > (p_VecY_G2, p_VecX2,
        p_MatA2, ELEM_PER_VECTOR);

    CUDA_CALL(hipGetLastError())

    //    // hipDeviceSynchronize waits for the kernel to finish, and returns
    //    // any errors encountered during the launch.
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(p_VecY_G, p_VecY_G2, size, hipMemcpyDeviceToHost))


        Error:
    hipFree(p_VecY_G2);
    hipFree(p_VecX2);
    hipFree(p_MatA2);
}

void init_data(int size) {
    srand((unsigned)time(NULL));
    FILE *fp = fopen("gen.bin", "wb");
    fwrite(&size, sizeof(int), 1, fp);

    int i, j;
    float x;

    for (i = 0; i < size; i++) {
        for (j = 0; j < ELEM_PER_VECTOR; j++) {
            x = 2.0f*((float)rand() / RAND_MAX) - 1.0f;
            fwrite(&x, sizeof(float), 1, fp);
        }
    }

    for (i = 0; i < ELEM_PER_VECTOR; i++) {
        for (j = 0; j < ELEM_PER_VECTOR; j++) {
            x = 2.0f*((float)rand() / RAND_MAX) - 1.0f;
            fwrite(&x, sizeof(float), 1, fp);
        }
    }

    fclose(fp);

    return;
}

int main()
{
    init_data(ELEMENT_SIZE);
    init_MatVec();
    printf("n = %d  file open ok.\n", n);

    CHECK_TIME_START;
    Mat_Vec_Multiply();
    CHECK_TIME_END(compute_time);

    printf("***CPU C[10] = %.3f/ Time taken = %.6fms\n", pVecY[0 * ELEM_PER_VECTOR + 0],
        compute_time);
      Mat_Vec_Multiply_GPU(pVecX, pMatA, pVecY_G);
//    Mat_Vec_Multiply_Kernel(pVecY, pVecX, pMatA, ELEM_PER_VECTOR);
    printf("***GPU C[10] = %.3f/ Time taken = %.6fms\n", pVecY_G[0 * ELEM_PER_VECTOR + 0],
        device_time);

    int vec_idx, i;

    for (i = 0; i < ELEMENT_SIZE * ELEM_PER_VECTOR; i++) {
        if (fabs(pVecY[i] - pVecY_G[i]) > 0.001) {
            printf("Kernel execution fail!!\n\n");
            break;
        }
    }
}

#endif