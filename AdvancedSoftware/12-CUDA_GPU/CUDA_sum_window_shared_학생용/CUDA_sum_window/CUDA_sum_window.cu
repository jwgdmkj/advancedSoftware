#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//kimyu7@sogang.ac.kr ���λ������̸��� �ص帳�ϴ�
#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#define	IN
#define OUT
#define INOUT

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define N_SIZE (1 << 26)													// ��ü ������ ������
#define NF_SIZE (1 << 6)													// Nf ũ��

#define NO_SHARED 0															// shared memory�� ������� �ʴ� Ŀ�� ���� flag
#define SHARED 1															// shared memory�� ����ϴ� Ŀ�� ���� flag

#define BLOCK_SIZE (1 << 6)													// CUDA Ŀ�� thread block ������

#define BLOCK_WIDTH (1 << 3)
#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)

#define N_ITERATION (1 << 0)												// ���� �ݺ� Ƚ��

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int Nf;

int *h_ArrayElements;
int *h_SumOfArrayElements_CPU;
int *h_SumOfArrayElements_GPU_No_Shared;
int *h_SumOfArrayElements_GPU_Shared;

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, int Nf, int Shared_flag);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� Ŀ�� �ڵ�
//	�� Ŀ���� shared �޸𸮸� ������� �ʴ´�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

extern __shared__ int shared_buffer[];

__global__ void Sum_n_elements_Kernel_No_shared(IN int *d_ArrayElements, OUT int *d_SumOfArrayElements,
	int N, int Nf) {
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	int sum = 0;

	for (int i = -Nf; i <= Nf; i++) {
		if (id + i >= N || id + i < 0) continue;
		sum += d_ArrayElements[id + i];
	}

	/*
	for (i = 0; i < N; i++) {
		sum = 0;
		for (j = -Nf; j <= Nf; j++) {
			if (i + j >= N || i + j < 0) continue;
			sum += p_ArrayElements[i + j];
		}
		p_SumOfElements_CPU[i] = sum;
	}
	*/
	d_SumOfArrayElements[id] = sum;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� Ŀ�� �ڵ�
//	�� Ŀ���� shared �޸𸮸� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/*
	Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_No_Shared, Nf, NO_SHARED)
	Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_Shared, Nf, SHARED);

	sizeof(int) * (BLOCK_SIZE+ 2*Nf)
	*/

__global__ void Sum_n_elements_Kernel_shared(IN int *d_ArrayElements,
	OUT int *d_SumOfArrayElements, int N, int Nf) {
	const unsigned block_id	 = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	/*Todo*/
	int i;
	int sum = 0;
	//�۷ι��޸𸮳����� �ξ��޸𸮷� ī��
	if (thread_id == 0) {
		for (i = 0; i < Nf; i++) {
			if (id + i < Nf)
				shared_buffer[i] = 0;
			else
				shared_buffer[i] = d_ArrayElements[id + i - Nf];
		}
	}
	//threadIdx==blocksize-1�� ��쵵 ��������
	if (thread_id == BLOCK_SIZE - 1) {
		for (i = 0; i < Nf; i++) {
			if (id + i >= N-1)
				shared_buffer[i+BLOCK_SIZE+Nf] = 0;
			else
				shared_buffer[i + BLOCK_SIZE + Nf] = d_ArrayElements[id + i + 1];
		}
	}

	shared_buffer[thread_id + Nf] = d_ArrayElements[id];

	//��ũ���߰�, �ξ��޸� �����ؼ� sum���ϰ�, sum�� ����� output�޸𸮿� ����
	__syncthreads();


	for (i = 0; i <= 2*Nf; i++) {
//		if (id + i >= N || id + i < 0) continue;
		sum += shared_buffer[i+ thread_id];
	}

	d_SumOfArrayElements[id] = sum;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�迭�� index - Nf ���� index + Nf ������ ������ ���� ����ϴ� C �ڵ�
//	GPU kernel�� ����� �񱳸� ���� ���� ����� �Ͽ����� �Ǵ��ϴ� �����ͷ� Ȱ��
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Sum_n_elements_CPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_CPU, int Nf) {
	int i, j, sum;

	for (i = 0; i < N; i++) {
		sum = 0;
		for (j = -Nf; j <= Nf; j++) {
			if (i + j >= N || i + j < 0) continue;
			sum += p_ArrayElements[i + j];
		}
		p_SumOfElements_CPU[i] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	�־��� bin ������ �д� �ڵ�
//	ù 4����Ʈ�� ��ü �������� ����, ���� 4����Ʈ�� Nf�� ũ��, �� ���� N���� int�� �����Ͱ� ����
//	�����ʹ� -100 ~ 100 ������ ���� ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void read_bin_file() {
	printf("***Binary File Read Start!!\n");
	FILE *fp = fopen("gen.bin", "rb");
	fread(&N, sizeof(int), 1, fp);
	fread(&Nf, sizeof(int), 1, fp);

	h_ArrayElements = (int *)malloc(N * sizeof(int));
	h_SumOfArrayElements_CPU = (int *)malloc(N * sizeof(int));
	h_SumOfArrayElements_GPU_No_Shared = (int *)malloc(N * sizeof(int));
	h_SumOfArrayElements_GPU_Shared = (int *)malloc(N * sizeof(int));

	fread(h_ArrayElements, sizeof(int), N, fp);

	fclose(fp);
	printf("***Binary File Read End!!\n\n");
}

void init_bin_file(IN int n, IN int nf) {
	printf("***Binary File Create Start!!\n");
	srand((unsigned)time(NULL));
	FILE *fp = fopen("gen.bin", "wb");
	fwrite(&n, sizeof(int), 1, fp);
	fwrite(&nf, sizeof(int), 1, fp);

	int i, input;

	for (i = 0; i < n; i++) {
		input = (int)((float)rand() / RAND_MAX * 200 - 100);
		fwrite(&input, sizeof(int), 1, fp);
	}

	fclose(fp);
	printf("***Binary File Create End!!\n\n");
}

int main()
{
	int i;
	init_bin_file(N_SIZE, NF_SIZE);
	read_bin_file();

	TIMER_T CPU_time = 0.0f, GPU_time_NO_SHARED = 0.0f, GPU_time_SHARED = 0.0f;

	for (i = 0; i < N_ITERATION; i++) {
		CHECK_TIME_START;
		Sum_n_elements_CPU(h_ArrayElements, h_SumOfArrayElements_CPU, Nf);
		CHECK_TIME_END(compute_time);
		CPU_time += compute_time;

		Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_No_Shared, Nf, NO_SHARED);
		GPU_time_NO_SHARED += device_time;

		Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_Shared, Nf, SHARED);
		GPU_time_SHARED += device_time;
	}

	for (i = 0; i < N; i++) {
		if (h_SumOfArrayElements_CPU[i] != h_SumOfArrayElements_GPU_No_Shared[i]
			|| h_SumOfArrayElements_CPU[i] != h_SumOfArrayElements_GPU_Shared[i]) {
			printf("%d : CPU : %d,\tGPU no shared : %d,\tGPU shared : %d\n",
				i, h_SumOfArrayElements_CPU[i], h_SumOfArrayElements_GPU_No_Shared[i],
				h_SumOfArrayElements_GPU_Shared[i]);
			break;
		}
	}

	if (i == N)
		printf("***Kernel execution Success!!\n\n");

	printf("***CPU compute time : %.3f ms\n", CPU_time / N_ITERATION);
	printf("***GPU NO SHARED compute time : %.3f ms\n", GPU_time_NO_SHARED / N_ITERATION);
	printf("***GPU SHARED compute time : %.3f ms\n", GPU_time_SHARED / N_ITERATION);

	free(h_ArrayElements);
	free(h_SumOfArrayElements_CPU);
	free(h_SumOfArrayElements_GPU_No_Shared);
	free(h_SumOfArrayElements_GPU_Shared);

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, int Nf, int Shared_flag) {
	hipError_t cudaStatus;

	CUDA_CALL(hipSetDevice(0));

	int *d_ArrayElements, *d_SumOfElements;
	size_t mem_size;

	mem_size = N * sizeof(int);
	CUDA_CALL(hipMalloc(&d_ArrayElements, mem_size));
	CUDA_CALL(hipMalloc(&d_SumOfElements, mem_size));

	CUDA_CALL(hipMemcpy(d_ArrayElements, p_ArrayElements, mem_size, hipMemcpyHostToDevice));

	dim3 blockDIm(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 gridDim(N / BLOCK_SIZE); 
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();


	/*
    Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_No_Shared, Nf, NO_SHARED)
	Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_Shared, Nf, SHARED);

	sizeof(int) * (BLOCK_SIZE+ 2*Nf)
	*/
	switch (Shared_flag)
	{
	case NO_SHARED:
		Sum_n_elements_Kernel_No_shared << <gridDim, blockDIm >> > (d_ArrayElements, d_SumOfElements, N, Nf);
		break;
	case SHARED:
		//shared�޸� �󸶳� ������ ������� �κ�. shared�޸� �����Ҵ��� ����
		//cudabuffer, �� 3��° ���ڿ� �󸶳� �� �� �����. ���� �Լ����� ����
		Sum_n_elements_Kernel_shared << <gridDim, blockDIm, sizeof(int) * (BLOCK_SIZE+ 2*Nf) >> >
			(d_ArrayElements, d_SumOfElements, N, Nf);
		break;
	}

	CUDA_CALL(cudaStatus = hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_SumOfElements_GPU, d_SumOfElements, mem_size, hipMemcpyDeviceToHost));
	
	hipFree(d_ArrayElements);
	hipFree(d_SumOfElements);

	return cudaStatus;
}
